#include "hip/hip_runtime.h"
#include "graph_harnar.h"

#include <hip/hip_runtime.h>
#include <>

#include "cudamem.h"
#include "util.h"

bool GraphHarnar::CudaInitialized = false;

int GraphHarnar::add_vertex()
{
	int value = Graph::add_vertex();
	this->dirty = true;

	return value;
}
void GraphHarnar::add_edge(int from, int to, unsigned int cost)
{
	Graph::add_edge(from, to, cost);
	this->dirty = true;
}

void GraphHarnar::initCuda()
{
	if (!GraphHarnar::CudaInitialized)
	{
		hipSetDeviceFlags(hipDeviceMapHost);
		GraphHarnar::CudaInitialized = true;
	}
}

__global__ void bfsKernel(LinearizedVertexHarnar* vertices, Edge* edges, size_t size)
{
	int offset = (blockDim.x * blockDim.y) * blockIdx.x;	// how many blocks skipped
	int blockPos = blockDim.x * threadIdx.y + threadIdx.x;	// position in block
	int pos = offset + blockPos;

	if (pos >= size) return;

	if (vertices[pos].frontier)
	{
		vertices[pos].frontier = false;

		int edgeCount = vertices[pos].edgeCount;
		int edgeIndex = vertices[pos].edgeIndex;

		for (size_t i = 0; i < edgeCount; i++)
		{
			int edge = edges[edgeIndex + i].target;
			if (!vertices[edge].visited)
			{
				vertices[edge].frontier_next = true;
			}
		}
	}
}
__global__ void bfsRequeueKernel(LinearizedVertexHarnar* vertices, size_t size, int target, bool *stop)
{
	int offset = (blockDim.x * blockDim.y) * blockIdx.x;	// how many blocks skipped
	int blockPos = blockDim.x * threadIdx.y + threadIdx.x;	// position in block
	int pos = offset + blockPos;

	if (pos >= size) return;

	if (vertices[pos].frontier_next)
	{
		vertices[pos].frontier = true;
		vertices[pos].frontier_next = false;
		vertices[pos].visited = true;

		stop[0] = false;

		if (pos == target)
		{
			stop[1] = true;
		}
	}
}
bool GraphHarnar::is_connected(int from, int to)
{
	if (!this->has_vertex(from) || !this->has_vertex(to)) return false;

	this->relinearizeVertices();
	this->initCuda();

	if (this->edges.size() < 1) return false;

	int graphSize = (int) this->vertices.size();

	this->linearizedVertices[from].frontier = true;
	this->linearizedVertices[from].visited = true;

	CudaMemory<LinearizedVertexHarnar> verticesCuda(graphSize, &(this->linearizedVertices[0]));
	CudaMemory<Edge> edgesCuda(this->edges.size(), &(this->edges[0]));
	CudaHostMemory<bool> stopCuda(2);

	// computation
	dim3 blockDim(32, 32);
	int blockCount = (graphSize / (blockDim.x * blockDim.y)) + 1;
	dim3 gridDim(blockCount, 1);

	bool* stopHost = stopCuda.host();
	stopHost[0] = false;

	while (!stopHost[0])
	{
		stopHost[0] = true;

		bfsKernel << <gridDim, blockDim >> >(*verticesCuda, *edgesCuda, graphSize);
		hipDeviceSynchronize();
		bfsRequeueKernel << <gridDim, blockDim >> >(*verticesCuda, graphSize, to, stopCuda.device());
		hipDeviceSynchronize();

		if (stopHost[1])
		{
			return true;
		}
	}

	return false;
}

__global__ void dijkstraKernel(LinearizedVertexHarnar* vertices, Edge* edges, unsigned int* costs, unsigned int* nextCosts, size_t size)
{
	int offset = (blockDim.x * blockDim.y) * blockIdx.x;	// how many blocks skipped
	int blockPos = blockDim.x * threadIdx.y + threadIdx.x;	// position in block
	int pos = offset + blockPos;

	if (pos >= size) return;

	if (vertices[pos].frontier)
	{
		vertices[pos].frontier = false;
		unsigned int distance = costs[pos];

		for (size_t i = 0; i < vertices[pos].edgeCount; i++)
		{
			Edge& edge = edges[vertices[pos].edgeIndex + i];
			atomicMin(nextCosts + edge.target, distance + edge.cost);
		}
	}
}
__global__ void dijkstraRequeueKernel(LinearizedVertexHarnar* vertices, unsigned int* costs, unsigned int* nextCosts, size_t size, bool *stop)
{
	int offset = (blockDim.x * blockDim.y) * blockIdx.x;	// how many blocks skipped
	int blockPos = blockDim.x * threadIdx.y + threadIdx.x;	// position in block
	int pos = offset + blockPos;

	if (pos >= size) return;

	if (nextCosts[pos] < costs[pos])
	{
		vertices[pos].frontier = true;
		costs[pos] = nextCosts[pos];
		*stop = false;
	}

	nextCosts[pos] = costs[pos];

}
unsigned int GraphHarnar::get_shortest_path(int from, int to)
{
	if (!this->has_vertex(from) || !this->has_vertex(to)) return UINT_MAX;

	this->relinearizeVertices();
	this->initCuda();

	if (this->edges.size() < 1) return UINT_MAX;

	int graphSize = (int) this->vertices.size();

	this->linearizedVertices[from].frontier = true;

	CudaMemory<LinearizedVertexHarnar> verticesCuda(graphSize, &(this->linearizedVertices[0]));
	CudaMemory<Edge> edgesCuda(this->edges.size(), &(this->edges[0]));

	std::vector<unsigned int> costs(graphSize, UINT_MAX);
	CudaMemory<unsigned int> costsCuda(graphSize, 0xFF);
	CudaMemory<unsigned int> nextCostsCuda(graphSize, 0xFF);

	CudaHostMemory<bool> stopCuda;

	// computation
	costsCuda.store(0, 1, from);

	dim3 blockDim(32, 32);
	int blockCount = (graphSize / (blockDim.x * blockDim.y)) + 1;
	dim3 gridDim(blockCount, 1);

	bool* stopHost = stopCuda.host();
	*stopHost = false;

	while (!(*stopHost))
	{
		*stopHost = true;

		dijkstraKernel << <gridDim, blockDim >> >(*verticesCuda, *edgesCuda, *costsCuda, *nextCostsCuda, graphSize);
		hipDeviceSynchronize();
		dijkstraRequeueKernel << <gridDim, blockDim >> >(*verticesCuda, *costsCuda, *nextCostsCuda, graphSize, stopCuda.device());
		hipDeviceSynchronize();
	}

	costsCuda.load(costs[0], graphSize);

	return costs[to];
}

void GraphHarnar::relinearizeVertices()
{
	this->edges.clear();
	this->linearizedVertices.clear();

	for (const Vertex& vertex : this->vertices)
	{
		int edgeCount = (int)vertex.edges.size();
		int edgeIndex = (int)edges.size();

		this->edges.insert(this->edges.end(), vertex.edges.begin(), vertex.edges.end());
		this->linearizedVertices.emplace_back(edgeIndex, edgeCount);
	}

	this->dirty = false;
}