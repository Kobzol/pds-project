#include "hip/hip_runtime.h"
#include "graph_cuda.h"

#include <hip/hip_runtime.h>
#include <>

#include "cudamem.h"
#include "util.h"

bool GraphCUDA::CudaInitialized = false;

int GraphCUDA::add_vertex()
{
	int value = Graph::add_vertex();
	this->dirty = true;

	return value;
}
void GraphCUDA::add_edge(int from, int to, unsigned int cost)
{
	Graph::add_edge(from, to, cost);
	this->dirty = true;
}

void GraphCUDA::initCuda()
{
	if (!GraphCUDA::CudaInitialized)
	{
		hipSetDeviceFlags(hipDeviceMapHost);
		GraphCUDA::CudaInitialized = true;
	}
}

__global__ void bfsKernel(LinearizedVertex* vertices, Edge* edges, int visitCounter, int target, bool* stop, size_t size)
{
	int offset = (blockDim.x * blockDim.y) * blockIdx.x;	// how many blocks skipped
	int blockPos = blockDim.x * threadIdx.y + threadIdx.x;	// position in block
	int pos = offset + blockPos;

	if (pos >= size) return;

	if (vertices[pos].visitIndex == visitCounter)
	{
		vertices[pos].visitIndex = CUDA_VISITED;

		int edgeCount = vertices[pos].edgeCount;
		int edgeIndex = vertices[pos].edgeIndex;

		stop[0] = false;
		if (pos == target)
		{
			stop[1] = true;
		}

		for (size_t i = 0; i < edgeCount; i++)
		{
			int edge = edges[edgeIndex + i].target;

			if (vertices[edge].visitIndex != CUDA_VISITED)
			{
				vertices[edge].visitIndex = visitCounter + 1;
			}
		}
	}
}
bool GraphCUDA::is_connected(int from, int to)
{
	if (!this->has_vertex(from) || !this->has_vertex(to)) return false;

	this->relinearizeVertices(true);
	this->initCuda();

	if (this->edges.size() < 1) return false;

	int graphSize = (int) this->vertices.size();

	this->linearizedVertices[from].visitIndex = 0;

	CudaMemory<LinearizedVertex> verticesCuda(graphSize, &(this->linearizedVertices[0]));
	CudaMemory<Edge> edgesCuda(this->edges.size(), &(this->edges[0]));
	CudaHostMemory<bool> stopCuda(2);
	int visitCounter = 0;

	// computation
	dim3 blockDim(16, 16);
	int blockCount = (graphSize / (blockDim.x * blockDim.y)) + 1;
	dim3 gridDim(blockCount, 1);

	bool* stopHost = stopCuda.host();
	stopHost[0] = false;

	while (!stopHost[0])
	{
		stopHost[0] = true;

		bfsKernel << <gridDim, blockDim >> >(*verticesCuda, *edgesCuda, visitCounter, to, stopCuda.device(), graphSize);
		hipDeviceSynchronize();

		if (stopHost[1])
		{
			return true;
		}

		visitCounter++;
	}

	return false;
}

__global__ void dijkstraKernel(LinearizedVertex* vertices, Edge* edges, unsigned int* costs, int visitCounter, bool *stop, size_t size)
{
	int offset = (blockDim.x * blockDim.y) * blockIdx.x;	// how many blocks skipped
	int blockPos = blockDim.x * threadIdx.y + threadIdx.x;	// position in block
	int pos = offset + blockPos;

	if (pos >= size) return;

	if (vertices[pos].visitIndex == visitCounter)
	{
		vertices[pos].visitIndex = CUDA_NOT_VISITED;
		unsigned int distance = costs[pos];

		int edgeCount = vertices[pos].edgeCount;
		int edgeIndex = vertices[pos].edgeIndex;

		for (size_t i = 0; i < edgeCount; i++)
		{
			Edge& edge = edges[edgeIndex + i];
			unsigned int newDistance = distance + edge.cost;
			if (atomicMin(&costs[edge.target], newDistance) > newDistance)
			{
				stop[0] = false;
				vertices[edge.target].visitIndex = visitCounter + 1;
			}
		}
	}
}
unsigned int GraphCUDA::get_shortest_path(int from, int to)
{
	if (!this->has_vertex(from) || !this->has_vertex(to)) return UINT_MAX;

	this->relinearizeVertices();
	this->initCuda();

	if (this->edges.size() < 1) return UINT_MAX;

	int graphSize = (int) this->vertices.size();

	unsigned int visitCounter = 0;
	this->linearizedVertices[from].visitIndex = visitCounter;

	CudaMemory<LinearizedVertex> verticesCuda(graphSize, &(this->linearizedVertices[0]));
	CudaMemory<Edge> edgesCuda(this->edges.size(), &(this->edges[0]));

	CudaMemory<unsigned int> costsCuda(graphSize, 0xEE);
	CudaHostMemory<bool> stopCuda;

	// computation
	costsCuda.store(0, 1, from);

	dim3 blockDim(16, 16);
	int blockCount = (graphSize / (blockDim.x * blockDim.y)) + 1;
	dim3 gridDim(blockCount, 1);

	bool* stopHost = stopCuda.host();
	*stopHost = false;

	while (!(*stopHost))
	{
		*stopHost = true;

		dijkstraKernel << <gridDim, blockDim >> >(*verticesCuda, *edgesCuda, *costsCuda, visitCounter, stopCuda.device(), graphSize);
		hipDeviceSynchronize();

		visitCounter++;
	}

	std::vector<unsigned int> costs(graphSize);
	costsCuda.load(costs[0], graphSize);

	return costs[to];
}

void GraphCUDA::relinearizeVertices(bool force)
{
	if (this->dirty || force)
	{
		this->edges.clear();
		this->linearizedVertices.clear();

		for (const Vertex& vertex : this->vertices)
		{
			int edgeCount = (int)vertex.edges.size();
			int edgeIndex = (int)edges.size();

			this->edges.insert(this->edges.end(), vertex.edges.begin(), vertex.edges.end());
			this->linearizedVertices.emplace_back(edgeIndex, edgeCount);
		}
	}

	this->dirty = false;
}